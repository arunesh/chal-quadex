#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <cassert>
#include "cgbn/cgbn.h"
#include "utility/support.h"


#define TPI 32
#define BITS 768 

#define TPB 128    // the number of threads per block to launch (must be divisible by 32

typedef struct {
  cgbn_mem_t<BITS> x;
  cgbn_mem_t<BITS> y;
  cgbn_mem_t<BITS> m;
  cgbn_mem_t<BITS> mul_lo;
  cgbn_mem_t<BITS> mul_hi;
} my_instance_t;


typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, 768> env1024_t;

const uint64_t MNT4_INV = 0xf2044cfbe45e7fff;
const uint64_t MNT6_INV = 0xc90776e23fffffff;


// num is of size 2*n. modulus is of size n
// result is of size n.
void reduce_wide(mp_limb_t* result, mp_limb_t* num, mp_limb_t* modulus, uint64_t inv, int n) {
        mp_limb_t *res = num;
        // mp_limb_t res[2*n];
        // mpn_mul_n(res, this->mont_repr.data, other.data, n);

        /*
          The Montgomery reduction here is based on Algorithm 14.32 in
          Handbook of Applied Cryptography
          <http://cacr.uwaterloo.ca/hac/about/chap14.pdf>.
         */
        for (size_t i = 0; i < n; ++i)
        {
            mp_limb_t k = inv * res[i];
            /* calculate res = res + k * mod * b^i */
            mp_limb_t carryout = mpn_addmul_1(res+i, modulus, n, k);
            carryout = mpn_add_1(res+n+i, res+n+i, n-i, carryout);
            assert(carryout == 0);
        }

        if (mpn_cmp(res+n, modulus, n) >= 0)
        {
            const mp_limb_t borrow = mpn_sub(res+n, res+n, n, modulus, n);
            assert(borrow == 0);
        }

        mpn_copyi(result, res+n, n);
}

__device__
 void store_np0(env1024_t::cgbn_t& l, uint32_t np0) {
  #if defined(__CUDA_ARCH__)
  #warning "including limbs code"
   l._limbs[10] = np0;
   l._limbs[11] = 0xe45e7fffu;
   printf("one %x, np-0 = %x\n", l._limbs[10], l._limbs[11]);
  #endif
}

__global__ void my_kernel(my_instance_t *problem_instances, uint32_t instance_count) {
  context_t         bn_context;                                 // create a CGBN context
  env1024_t         bn1024_env(bn_context);                     // construct a bn environment for 1024 bit math
  env1024_t::cgbn_t a, b, m;                      // three 1024-bit values (spread across a warp)
  env1024_t::cgbn_wide_t mul_wide;
  // uint32_t np0;
  
  int32_t my_instance=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;  // determine my instance number
  
  if(my_instance>=instance_count) return;                         // return if my_instance is not valid
  
  cgbn_load(bn1024_env, a, &(problem_instances[my_instance]).x);
  cgbn_load(bn1024_env, b, &(problem_instances[my_instance]).y);
  cgbn_load(bn1024_env, m, &(problem_instances[my_instance]).m);

  // np0 = -cgbn_binary_inverse_ui32(bn1024_env, cgbn_get_ui32(bn1024_env, m));

  cgbn_mul_wide(bn1024_env, mul_wide, a, b);

  cgbn_store(bn1024_env, &(problem_instances[my_instance].mul_lo), mul_wide._low);
  cgbn_store(bn1024_env, &(problem_instances[my_instance].mul_hi), mul_wide._high);
}


void set_literal(cgbn_mem_t<BITS>& h, uint32_t literal, int num) {
  for (int i = 1; i < num; i ++ ) {
     h._limbs[i] = 0;
  }
  h._limbs[0] = literal;
}

void set_literal_limbs(cgbn_mem_t<BITS>& h, uint32_t literal, int num, int size) {
  for (int i = 0; i < num; i ++ ) {
     h._limbs[i] = literal;
  }
  for (int i = num; i < size; i ++ ) {
     h._limbs[i] = 0;
  }
}

void print_uint8_array(uint8_t* array, int size) {
    for (int i = 0; i < size; i ++) {
        printf("%02x", array[i]);
    }
    printf("\n");
}

std::vector<uint8_t*>* compute_newcuda(std::vector<uint8_t*> a, std::vector<uint8_t*> b, uint8_t* input_m_base, int num_bytes, uint64_t inv) {
  int num_elements = a.size();

  my_instance_t *gpuInstances;
  my_instance_t* instance_array = (my_instance_t*) malloc(sizeof(my_instance_t) * num_elements);
  cgbn_error_report_t *report;

  // create a cgbn_error_report for CGBN to report back errors
  NEW_CUDA_CHECK(cgbn_error_report_alloc(&report));
  for (int i = 0; i < num_elements; i ++) {
    std::memcpy((void*)instance_array[i].x._limbs, (const void*) a[i], num_bytes);
    std::memcpy((void*)instance_array[i].y._limbs, (const void*) b[i], num_bytes);
    std::memcpy((void*)instance_array[i].m._limbs, (const void*) input_m_base, num_bytes);
  }

  printf("Copying instances to the GPU ...\n");
  NEW_CUDA_CHECK(hipSetDevice(0));
  NEW_CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(my_instance_t)*num_elements));
  NEW_CUDA_CHECK(hipMemcpy(gpuInstances, instance_array, sizeof(my_instance_t)*num_elements, hipMemcpyHostToDevice));
  
  int tpb = TPB;
  printf("\n Threads per block =%d", tpb);
  int IPB = TPB/TPI;
  int tpi = TPI;
  printf("\n Threads per instance = %d", tpi);
  printf("\n Instances per block = %d", IPB);

  uint32_t num_blocks = (num_elements+IPB-1)/IPB;
  printf("\n Number of blocks = %d", num_blocks);

  my_kernel<<<num_blocks, TPB>>>(gpuInstances, num_elements);
  NEW_CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  printf("Copying results back to CPU ...\n");
  NEW_CUDA_CHECK(hipMemcpy(instance_array, gpuInstances, sizeof(my_instance_t)*num_elements, hipMemcpyDeviceToHost));


  int num_limbs = num_bytes / 8;
  printf("\n Setting num 64 limbs = %d", num_limbs);
  mp_limb_t* num = (mp_limb_t*)malloc(sizeof(mp_limb_t) * num_limbs * 2);
  mp_limb_t* modulus = (mp_limb_t*)malloc(sizeof(mp_limb_t) * num_limbs);
  std::memcpy((void*) modulus, (const void*) instance_array->m._limbs, num_bytes);

  printf("\n Dumping modulus:");
  gmp_printf("%Nx\n", modulus, num_limbs); 

  std::vector<uint8_t*>* res_vector = new std::vector<uint8_t*>();
  for (int i = 0; i < num_elements; i ++) {
    // Reduce
    std::memcpy((void*)num, (const void*)instance_array[i].mul_lo._limbs, num_bytes);
    std::memcpy((void*) (num + num_limbs), (const void*)instance_array[i].mul_hi._limbs, num_bytes);
    mp_limb_t* fresult = (mp_limb_t*)malloc(sizeof(mp_limb_t) * num_limbs);
 
    // printf("\n Dumping 64 byte limb wide num [%d]:", i);
    // gmp_printf("%Nx\n", num, num_limbs * 2); 

    reduce_wide(fresult, num, modulus, inv, num_limbs);

    // store the result.
    res_vector->emplace_back((uint8_t*)fresult);
  }
  free(num);
  free(modulus);
  free(instance_array);
  hipFree(gpuInstances);
  return res_vector;
}


std::pair<std::vector<uint8_t*>, std::vector<uint8_t*> > 
compute_quadex_cuda(std::vector<uint8_t*> x0_a0,
                    std::vector<uint8_t*> x0_a1,
                    std::vector<uint8_t*> y0_a0,
                    std::vector<uint8_t*> y0_a1,
                    uint8_t* input_m_base, int num_bytes, uint64_t inv) {
  int num_elements = x0_a0.size();
  std::pair<std::vector<uint8_t*>, std::vector<uint8_t*> > res;
  return res;
}
